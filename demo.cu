#include "hip/hip_runtime.h"
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <cassert>

template<typename RandomAccessIterator, typename BinaryFunction>
inline __device__
void blockwise_inplace_inclusive_scan(RandomAccessIterator first, BinaryFunction op)
{
  typename thrust::iterator_value<RandomAccessIterator>::type x = first[threadIdx.x];

  for(unsigned int offset = 1; offset < blockDim.x; offset *= 2)
  {
    if(threadIdx.x >= offset)
    {
      x = op(first[threadIdx.x - offset], x);
    }

    __syncthreads();

    first[threadIdx.x] = x;

    __syncthreads();
  }
}


template<typename RandomAccessIterator, typename Size, typename BinaryFunction>
inline __device__ 
void blockwise_inplace_small_inclusive_scan(RandomAccessIterator first, Size n, BinaryFunction op)
{
  typename thrust::iterator_value<RandomAccessIterator>::type x;

  if(threadIdx.x < n)
  {
    x = first[threadIdx.x];
  }

  for(Size offset = 1; offset < n; offset *= 2)
  {
    if(threadIdx.x >= offset)
    {
      x = op(first[threadIdx.x - offset], x);
    }

    __syncthreads();

    first[threadIdx.x] = x;

    __syncthreads();
  }
}


template<typename RandomAccessIterator, typename Size, typename BinaryFunction>
inline __device__ 
void blockwise_inplace_inclusive_scan(RandomAccessIterator first, Size n, BinaryFunction op)
{
  blockwise_inplace_small_inclusive_scan(first, min(blockDim.x, n), op);

  RandomAccessIterator last = first + n;
  for(first += blockDim.x; first < last; first += blockDim.x, n -= blockDim.x)
  {
    // sum the previous iteration's carry
    if(threadIdx.x == 0)
    {
      *first = op(*(first-1), *first);
    }

    __syncthreads();

    blockwise_inplace_small_inclusive_scan(first, min(blockDim.x, n), op);
  }
}


__global__ void inplace_scan(int *x, int n)
{
  blockwise_inplace_inclusive_scan(x, n, thrust::plus<int>());
}


int main()
{
  thrust::host_vector<size_t> sizes;
  sizes.push_back(0);
  sizes.push_back(1);
  sizes.push_back(9);
  sizes.push_back(31);
  sizes.push_back(32);
  sizes.push_back(33);
  sizes.push_back(512);
  sizes.push_back(1024 + 1);
  sizes.push_back(1 << 20);
  sizes.push_back(16 << 20);

  for(int block_size = 32; block_size <= 512; block_size += 32)
  {
    std::cout << "testing block_size " << block_size << std::endl;

    for(int i = 0; i < sizes.size(); ++i)
    {
      size_t n = sizes[i];

      std::cout << "testing size " << n << std::endl;

      thrust::device_vector<int> vec1(n, 1), vec2(n, 1);

      thrust::inclusive_scan(vec1.begin(), vec1.end(), vec1.begin());

      inplace_scan<<<1,block_size>>>(vec2.data().get(), vec2.size());

      if(n < 50)
      {
        std::cout << "result: ";
        thrust::copy(vec2.begin(), vec2.end(), std::ostream_iterator<int>(std::cout, " "));
        std::cout << std::endl;
      }

      assert(vec1 == vec2);
    }
  }

  return 0;
}

